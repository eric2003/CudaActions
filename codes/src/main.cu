
#include <hip/hip_runtime.h>
#include <cstdio>

int main(int argc, char* argv[])
{
    int num_gpus = 0;    
    hipGetDeviceCount( &num_gpus );
    
    std::printf("CUDA : hipGetDeviceCount : number of CUDA devices:\t%d\n", num_gpus);

    return 0;
}